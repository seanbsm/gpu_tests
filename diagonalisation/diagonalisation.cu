#include "hip/hip_runtime.h"

/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include syevd_example.cpp 
 *   g++ -o a.out syevd_example.o -L/usr/local/cuda/lib64 -lcudart -lcusolver
 *
 */
#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

/* Time-keeping modules */
#include <chrono>
#include <ctime>

#include "eigenFinder.h"

#include "dsyevd_stream_functions.cuh"
#include "dsyevj_stream_functions.cuh"
#include "dsyevj_batch_functions.cuh"
#include "kernel_functions.cuh"
#include "fill_matrices.cuh"

//~ typedef float  floatType;
typedef double floatType;

void printMatrix(int m, int n, const floatType*A, const char* name){
	for(int row = 0 ; row < m ; row++){
		for(int col = 0 ; col < n ; col++){
			floatType Areg = A[row + col*m];
			printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
		}
	}
}

int main(int argc, char*argv[]){
	
	const int N = 32;
	const int m = N;
	const int batchSize = 254*4;

	/* Declare host arrays */
	floatType *A = new floatType [m*m*batchSize];
	floatType *V = new floatType [m*m*batchSize];
	floatType *W = new floatType [m*batchSize];
	
	/* Declare device arrays */
	floatType *d_A  = NULL; /* m-by-m-by-batchSize */
	floatType *d_W  = NULL; /* m-by-batchSizee */
	
	/* Fill up array A with matrix elements */
	fillSymmetricMatrices_full(A, m, batchSize);
	
	/* Allocate A on device */
	hipMalloc ((void**)&d_A   , sizeof(floatType) * m * m * batchSize);
	hipMalloc ((void**)&d_W   , sizeof(floatType) * m * batchSize);
	
	/* Copy A to device */
	hipMemcpy(d_A, A, sizeof(floatType) * m * m * batchSize, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	 
	/* Call diagonalisation routine of choice */
	//~ double time_gpu = diagonalise_kernel(d_A, d_W, m, batchSize);
	double time_gpu = diagonalise_stream_syevd(d_A, d_W, m, batchSize);
	//~ double time_gpu = diagonalise_stream_syevj(d_A, d_W, m, batchSize);
	//~ double time_gpu = diagonalise_batch_syevj(d_A, d_W, m, batchSize);
	
	std::cout<<"Time gpu: "<< time_gpu << " s" << std::endl;
	
	/* Code A and W from device */
	hipMemcpy(V, d_A, sizeof(floatType) * m * m * batchSize, hipMemcpyDeviceToHost);
	hipMemcpy(W, d_W, sizeof(floatType) * m * batchSize      , hipMemcpyDeviceToHost);
	
	
	/* CPU BENCHMARKING */
	/* Symmetric matrix size */
	int matSize = m*(m+1)/2;
	
	/* Declare CPU arrays */
	double *A_CPU = new double [matSize*batchSize];
	double *V_CPU = new double [m*m*batchSize];
	double *W_CPU = new double [m*batchSize];
	
	/* Fill up array A with matrix elements */
	fillSymmetricMatrices_symm(A_CPU, m, batchSize);
	
	auto start = std::chrono::system_clock::now();
	
	/* Diagonalise A using MKL LAPACK */
	/* Arg 1 is A and must be upper-triangle */
	/* Arg 2 are the eigenvalues */
	/* Arg 3 are the eigenvectors */
	/* Arg 4 is the dimension */
	for (int i=0; i<batchSize; i++){
		findEigenReal(&A_CPU[i*matSize], &W_CPU[m*i], &V_CPU[i*m*m], m);
	}
	
	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> time_raw = end-start;
	double time_cpu = time_raw.count();
	
	std::cout<<"Time cpu: "<< time_cpu << " s" << std::endl;
	
	//~ for (int M=0; M<batchSize; M++){
		//~ for (int i=0; i<m; i++){
			//~ for (int j=0; j<m; j++){
				//~ std::cout << V_CPU[M*m*m + i*m+j] << std::endl;
			//~ }
		//~ }
	//~ }
	
	/* Print any numerically big differences in eigenvalues between GPU and CPU*/
	floatType maxDiff = 0;
	for (int M=0; M<batchSize; M++){
		for (int i=0; i<m; i++){
			floatType diff = abs(W_CPU[M*m + i] - W[M*m + i]);
			if (diff > maxDiff){
				maxDiff = diff;
				//~ std::cout << M << " " << i << " " << diff << std::endl;
			}
		}
	}
	
	std::cout << std::endl;
	std::cout << "Max diff: " << maxDiff << std::endl;

	/* free resources */
	if (d_A) hipFree(d_A);
	if (d_W) hipFree(d_W);

	delete [] A;
	delete [] V;
	delete [] W;

	delete [] A_CPU;
	delete [] V_CPU;
	delete [] W_CPU;

	hipDeviceReset();

	return 0;
}
