#include "hip/hip_runtime.h"

#include "dsyevj_batch_functions.cuh"

/* Uses cuda batch routines to diagonalise using Jacobi method.
 * NOTE: DUE TO SHARED MEMORY, THIS FUNCTION IS LIMITIED TO MAX 32X32
 * MATRICES (NO LIMIT ON BATCHSIZE) */
double diagonalise_batch_syevj(float  *d_A, float  *d_W, int m, int batchSize){
	
	hipsolverHandle_t cusolverH = NULL;
	hipStream_t stream			 = NULL;
	hipsolverSyevjInfo_t syevj_params	 = NULL;
	
	int* d_info    = NULL; 		/* batchSize */
	int lwork 	   = 0;  		/* size of workspace */
	float *d_work = NULL; 		/* device workspace for syevjBatched */
	
	const float tol = 1.e-7;
	const int max_sweeps = 20;
	const int sort_eig  = 1;   /* don't sort eigenvalues */
	const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */
	const hipblasFillMode_t  uplo = HIPBLAS_FILL_MODE_UPPER;
	
	/* step 1: create cusolver handle, bind a stream  */
	hipsolverDnCreate(&cusolverH);
	
	hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	hipsolverSetStream(cusolverH, stream);
	
	/* step 2: configuration of syevj */
	hipsolverDnCreateSyevjInfo(&syevj_params);
	
	/* default value of tolerance is machine zero */
	hipsolverDnXsyevjSetTolerance(syevj_params, tol);

	/* default value of max. sweeps is 100 */
	hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps);
	
	/* disable sorting */
	hipsolverDnXsyevjSetSortEig( syevj_params, sort_eig);
	
	hipMalloc ((void**)&d_info, sizeof(int) * batchSize);
	
	//~ auto start = std::chrono::system_clock::now();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	//~ hipDeviceSynchronize();
	
	/* step 4: query working space of syevjBatched */
	hipsolverDnSsyevjBatched_bufferSize(cusolverH,
									   jobz,
									   uplo,
									   m,
									   d_A,
									   m,
									   d_W,
									   &lwork,
									   syevj_params,
									   batchSize);
	
	hipMalloc((void**)&d_work, sizeof(float)*lwork);
	
	/* step 5: compute spectrum of A0 and A1 */
	hipsolverDnSsyevjBatched(cusolverH,
							jobz,
							uplo,
							m,
							d_A,
							m,
							d_W,
							d_work,
							lwork,
							d_info,
							syevj_params,
							batchSize);
	/* Without this synchronisation I get crazy eigenvalues in my print test */
	hipDeviceSynchronize();
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	//~ if (d_info ) hipFree(d_info);
	//~ if (d_work ) hipFree(d_work);

	//~ if (cusolverH) hipsolverDnDestroy(cusolverH);
	//~ if (stream      ) hipStreamDestroy(stream);
	//~ if (syevj_params) hipsolverDnDestroySyevjInfo(syevj_params);

	return (double) time*1e-3;
}

/* Uses cuda batch routines to diagonalise using Jacobi method.
 * NOTE: DUE TO SHARED MEMORY, THIS FUNCTION IS LIMITIED TO MAX 32X32
 * MATRICES (NO LIMIT ON BATCHSIZE) */
double diagonalise_batch_syevj(double *d_A, double *d_W, int m, int batchSize){
	
	hipsolverHandle_t cusolverH = NULL;
	hipStream_t stream			 = NULL;
	hipsolverSyevjInfo_t syevj_params	 = NULL;
	
	int* d_info    = NULL; 		/* batchSize */
	int lwork 	   = 0;  		/* size of workspace */
	double *d_work = NULL; 		/* device workspace for syevjBatched */
	
	const double tol = 1.e-15;
	const int max_sweeps = 20;
	const int sort_eig  = 1;   /* don't sort eigenvalues */
	const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */
	const hipblasFillMode_t  uplo = HIPBLAS_FILL_MODE_UPPER;
	
	/* step 1: create cusolver handle, bind a stream  */
	hipsolverDnCreate(&cusolverH);
	
	hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	hipsolverSetStream(cusolverH, stream);
	
	/* step 2: configuration of syevj */
	hipsolverDnCreateSyevjInfo(&syevj_params);
	
	/* default value of tolerance is machine zero */
	hipsolverDnXsyevjSetTolerance(syevj_params, tol);

	/* default value of max. sweeps is 100 */
	hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps);
	
	/* disable sorting */
	hipsolverDnXsyevjSetSortEig( syevj_params, sort_eig);
	
	hipMalloc ((void**)&d_info, sizeof(int) * batchSize);
	
	//~ auto start = std::chrono::system_clock::now();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	//~ hipDeviceSynchronize();
	
	/* step 4: query working space of syevjBatched */
	hipsolverDnDsyevjBatched_bufferSize(cusolverH,
									   jobz,
									   uplo,
									   m,
									   d_A,
									   m,
									   d_W,
									   &lwork,
									   syevj_params,
									   batchSize);
	
	hipMalloc((void**)&d_work, sizeof(double)*lwork);
	
	/* step 5: compute spectrum of A0 and A1 */
	hipsolverDnDsyevjBatched(cusolverH,
							jobz,
							uplo,
							m,
							d_A,
							m,
							d_W,
							d_work,
							lwork,
							d_info,
							syevj_params,
							batchSize);
	/* Without this synchronisation I get crazy eigenvalues in my print test */
	hipDeviceSynchronize();
	
	//~ auto end = std::chrono::system_clock::now();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	//~ if (d_info ) hipFree(d_info);
	//~ if (d_work ) hipFree(d_work);

	//~ if (cusolverH) hipsolverDnDestroy(cusolverH);
	//~ if (stream      ) hipStreamDestroy(stream);
	//~ if (syevj_params) hipsolverDnDestroySyevjInfo(syevj_params);
	
	return (double) time*1e-3;
}
