#include "hip/hip_runtime.h"

#include "jacobi_kernel_function_new.cuh"

/* Symmetric 2-by-2 Schur decomposition */
__global__
void sym_Schur2_all(floatType *d_A, floatType *d_c, floatType *d_s, int *d_top, int *d_bot, int n){
	
	int h = blockIdx.x;
	int k = threadIdx.x;
	
	floatType *A = &d_A[h*n*n];
	floatType *C = &d_c[h*n/2];
	floatType *S = &d_s[h*n/2];
	
	int tk = d_top[k];
	int bk = d_bot[k];
	int p = (tk<bk)*tk + (tk>bk)*bk;
	int q = (tk>bk)*tk + (tk<bk)*bk;
	
	floatType tau, t, c, s, Apq, App, Aqq;
	
	Apq = A[n*p + q];
	App = A[n*p + p];
	Aqq = A[n*q + q];
	
	if ( Apq!=0 ){
		tau = (Aqq - App) / (2.*Apq);
		
		if (tau>=0){
			t =  1. / (tau + sqrt(1+tau*tau));
		}
		else{
			t = -1. / (-tau + sqrt(1+tau*tau));
		}
		
		c = 1. / sqrt(1+t*t);
		s = t*c;
	}
	else{
		c = 1;
		s = 0;
	}
	
	C[k] = c;
	S[k] = s;
}

__global__
void Jacobi_parallel_row_rot(floatType *d_A, floatType *d_V, floatType *d_c, floatType *d_s, int *d_top, int *d_bot, int n){
	
	int h = blockIdx.x;
	int K = threadIdx.x;
	
	int k = K / (n-1);
	int i = K % (n-1);
	
	/* Only usage of h and k */
	floatType *A = &d_A[h*n*n];
	floatType *V = &d_V[h*n*n];
	int tk 		 = d_top[k];
	int bk 		 = d_bot[k];
	
	floatType *c_set  = &d_c[h*n/2];
	floatType *s_set  = &d_s[h*n/2];
	floatType c = c_set[k];
	floatType s = s_set[k];
	
	/* Set p to the smallest of tk and bk */
	int p = (tk<bk)*tk + (tk>bk)*bk;
	
	/* Set q to the largest of tk and bk */
	int q = (tk>bk)*tk + (tk<bk)*bk;
	
	floatType Api, Aqi, App, Apq, Aqp, Aqq;
	
	if (i>=p){
		i += 1;
		
		if (i>=q){
			i += 1;
		}
	}
	
	/* TEMP SOLUTION */
	if (i==n){
		App = A[n*p + p];
		Apq = A[n*p + q];
		Aqp = A[n*q + p];
		Aqq = A[n*q + q];
		
		A[n*p + p] = c*c*App - c*s*(Apq + Aqp) + s*s*Aqq;
		A[n*q + q] = s*s*App + c*s*(Apq + Aqp) + c*c*Aqq;
		
		A[n*p + q] = 0;
		A[n*q + p] = 0;
	}
	else{
		Api = A[n*p + i];
		Aqi = A[n*q + i];
		
		A[n*p + i] = c*Api - s*Aqi;
		A[n*q + i] = c*Aqi + s*Api;
	}
}


__global__
void Jacobi_parallel_vec_rot(floatType *d_V, floatType *d_c, floatType *d_s, int *d_top, int *d_bot, int n){
	int h = blockIdx.x;
	int K = threadIdx.x;
	
	int k = K / n;
	int i = K % n;
	
	/* Only usage of h and k */
	floatType *V = &d_V[h*n*n];
	int tk 		 = d_top[k];
	int bk 		 = d_bot[k];
	
	floatType *c_set  = &d_c[h*n/2];
	floatType *s_set  = &d_s[h*n/2];
	floatType c = c_set[k];
	floatType s = s_set[k];
	
	/* Set p to the smallest of tk and bk */
	int p = (tk<bk)*tk + (tk>bk)*bk;
	
	/* Set q to the largest of tk and bk */
	int q = (tk>bk)*tk + (tk<bk)*bk;
	
	floatType Viq, Vip;
	
	Vip = V[i*n + p];
	Viq = V[i*n + q];
	
	V[i*n + p] = c*Vip - s*Viq;
	V[i*n + q] = c*Viq + s*Vip;
}


__global__
void Jacobi_parallel_col_rot(floatType *d_A, floatType *d_V, floatType *d_c, floatType *d_s, int *d_top, int *d_bot, int n){
	int h = blockIdx.x;
	int K = threadIdx.x;
	
	int k = K / (n/2 - 1);
	int i = K % (n/2 - 1);
	
	if (i>=k){
		i += 1;
	}
	
	/* Only usage of h and k */
	floatType *A 	   = &d_A[h*n*n];
	int tk 		   = d_top[k];
	int bk 		   = d_bot[k];
	
	floatType *c_set  = &d_c[h*n/2];
	floatType *s_set  = &d_s[h*n/2];
	
	/* Set p to the smallest of tk and bk */
	int p = (tk<bk)*tk + (tk>bk)*bk;
	/* Set q to the largest of tk and bk */
	int q = (tk>bk)*tk + (tk<bk)*bk;
	
	int ti = d_top[i];
	int bi = d_bot[i];
	
	int p_i = (ti<bi)*ti + (ti>bi)*bi;
	int q_i = (ti>bi)*ti + (ti<bi)*bi;
	
	floatType c = c_set[i];
	floatType s = s_set[i];
	
	floatType Ap_pi = A[n*p + p_i];
	floatType Aq_pi = A[n*q + p_i];
	
	floatType Ap_qi = A[n*p + q_i];
	floatType Aq_qi = A[n*q + q_i];
	
	A[n*p + p_i] = c*Ap_pi - s*Ap_qi;
	A[n*q + p_i] = c*Aq_pi - s*Aq_qi;
	
	A[n*p + q_i] = c*Ap_qi + s*Ap_pi;
	A[n*q + q_i] = c*Aq_qi + s*Aq_pi;
}

__global__
void update_A_prev(floatType *d_A, floatType *d_A_prev){
	int idx = threadIdx.x;
	
	d_A_prev[idx] = d_A[idx];
}

__global__
void rotational_sets_copy(int *top_new, int *bot_new, int *top, int *bot){
	int k = threadIdx.x;
	
	top_new[k] = top[k];
	bot_new[k] = bot[k];
}

__global__
void rotational_sets(int *top_new, int *bot_new, int *top, int *bot, int N){
	int k = threadIdx.x;
	int m = N/2;
	
	if (k==0){
		top_new[0] = 0;
	}
	else if (k==1){
		top_new[k] = bot[0];
	}
	else{
		top_new[k] = top[k-1];
	}
	
	if (k==m-1){
		bot_new[k] = top[k];
	}
	else{
		bot_new[k] = bot[k+1];
	}
}

double jacobi_kernels_parallel(floatType *d_A, floatType *d_W, int m, int batchSize){
	
	int nBlocks = batchSize;
	int nThreads = m/2;
	
	//~ const floatType eps = 1e-10;
	int maxIter = m+1;
	int m_half = m/2;
	
	int top [m_half];
	int bot [m_half];
	/* Initialise top and bottom indices */
	for (int i=0; i<m_half; i++){
		top[i] = 2*i;
		bot[i] = 2*i+1;
	}
	
	/* Declare rotation set device arrays */
	int *d_top = NULL; int *d_top_temp = NULL;
	int *d_bot = NULL; int *d_bot_temp = NULL;
	
	/* Allocate rotation set device arrays */
	hipMalloc( (void**)&d_top, m_half*sizeof(int) );
	hipMalloc( (void**)&d_bot, m_half*sizeof(int) );
	hipMalloc( (void**)&d_top_temp, m_half*sizeof(int) );
	hipMalloc( (void**)&d_bot_temp, m_half*sizeof(int) );
	
	/* Copy rotation set host arrays to corresponding device arrays */
	hipMemcpy(d_top, top, sizeof(int) * m_half, hipMemcpyHostToDevice);
	hipMemcpy(d_bot, bot, sizeof(int) * m_half, hipMemcpyHostToDevice);
	
	/* Declare rotation angle-set device arrays */
	floatType *d_c = NULL;
	floatType *d_s = NULL;
	hipMalloc( (void**)&d_c,  batchSize*m_half*sizeof(floatType) );
	hipMalloc( (void**)&d_s,  batchSize*m_half*sizeof(floatType) );
	
	
	floatType *d_V = NULL;
	hipMalloc( (void**)&d_V,  batchSize*m*m*sizeof(floatType) );
	
	floatType V [m*m*batchSize];
	for (int h=0; h<batchSize; h++){
		for (int i=0; i<m; i++){
			V[h*m*m + i*m + i] = 1;
		}
	}
	hipMemcpy(d_V, V, sizeof(floatType) * m*m*batchSize, hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	/* Loop over number of rotations */
	for (int iter=0; iter<maxIter; iter++){
		
		/* Loop over rotation sets */
		for (int j=0; j<m-1; j++){
			/* Calculate all rotation angles before we start rotating */
			sym_Schur2_all<<<nBlocks, nThreads>>>(d_A, d_c, d_s, d_top, d_bot, m);
			
			Jacobi_parallel_row_rot<<<nBlocks, nThreads * (m-1)>>>(d_A, d_V, d_c, d_s, d_top, d_bot, m);
			Jacobi_parallel_col_rot<<<nBlocks, nThreads*(nThreads-1)>>>(d_A, d_V, d_c, d_s, d_top, d_bot, m);
			
			Jacobi_parallel_vec_rot<<<nBlocks, nThreads*m>>>(d_V, d_c, d_s, d_top, d_bot, m);
			
			rotational_sets_copy<<<1, nThreads>>>(d_top_temp, d_bot_temp, d_top, d_bot);
			rotational_sets<<<1, nThreads>>>(d_top, d_bot, d_top_temp, d_bot_temp, m);
		}
	}
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	floatType A [m*m*batchSize];
	floatType W [m*batchSize];
	
	hipMemcpy(A, d_A, sizeof(floatType) * m*m*batchSize, hipMemcpyDeviceToHost);
	hipMemcpy(V, d_V, sizeof(floatType) * m*m*batchSize, hipMemcpyDeviceToHost);
	
	for (int h=0; h<batchSize; h++){
		for (int i=0; i<m; i++){
			W[m*h + i] = A[m*m*h + m*i + i];
		}
		for (int i=0; i<m*m; i++){
			A[m*m*h + i] = V[m*m*h + i];
		}
	}
	
	hipMemcpy(d_A, A, sizeof(floatType) * m*m*batchSize, hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, sizeof(floatType) * m*batchSize, hipMemcpyHostToDevice);
	
	return (double) time*1e-3;
}
