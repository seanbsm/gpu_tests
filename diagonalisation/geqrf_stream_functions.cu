
#include "geqrf_stream_functions.cuh"

double diagonalise_stream_geqrf(float  *d_A, float  *d_W, int m, int batchSize){
	
	int NBSTREAM = batchSize;
	
	hipsolverHandle_t cusolverH;
	hipblasHandle_t 	   cublasH;
	
	hipsolverDnCreate(&cusolverH);
	hipblasCreate(&cublasH);
	
	const double one = 1;
	
	//~ hipStream_t 	   stream	 [NBSTREAM];
	hipStream_t *stream = (hipStream_t *)malloc(NBSTREAM*sizeof(hipStream_t));
	
	int 	l_work [NBSTREAM];
	float  *d_work [NBSTREAM];
	int    *d_info [NBSTREAM];
	float  *d_tau  [NBSTREAM];
	
	for (int i=0; i<NBSTREAM; i++){
		hipMalloc((void **)&d_info[i], sizeof(int));
		hipMalloc ((void**)&d_tau[i],  sizeof(double)*m);
	}
	
	for (int i=0; i<NBSTREAM; i++){
		//~ hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
		hipStreamCreate(&stream[i]);
	}
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	//~ for(int i=0;i<NBSTREAM;i++){
		//~ hipsolverDnSgeqrf_bufferSize(cusolverH, 
									//~ m, 
									//~ m, 
									//~ &d_A[i*m*m], 
									//~ m, 
									//~ &l_work[i]);
	
		//~ hipMalloc((void **)&d_work[i], sizeof(double)*l_work[i]);
	//~ }
	
	//~ for(int i=0;i<NBSTREAM;i++){
		//~ hipsolverSetStream(cusolverH, stream[i]);
		
		//~ hipsolverDnSgeqrf(cusolverH, 
						 //~ m, 
						 //~ m, 
						 //~ &d_A[i*m*m], 
						 //~ m, 
						 //~ d_tau[i*m], 
						 //~ d_work[i], 
						 //~ l_work[i], 
						 //~ d_info[i]);
	//~ }
	
	//~ for(int i=0;i<NBSTREAM;i++){
		//~ hipsolverSetStream(cusolverH, stream[i]);
		
		//~ hipsolverDnSormqr(cusolverH, 
						 //~ HIPBLAS_SIDE_LEFT, 
						 //~ HIPBLAS_OP_T,
						 //~ m, 
						 //~ m, 
						 //~ m, 
						 //~ &d_A[i*m*m], 
						 //~ m,
						 //~ d_tau[i*m],
						 //~ &d_B[i*m*m],
						 //~ m,
						 //~ d_work[i],
						 //~ l_work[i],
						 //~ d_info[i]);
	//~ }
	
	//~ for(int i=0;i<NBSTREAM;i++){
		//~ hipsolverSetStream(cusolverH, stream[i]);
		
		//~ hipblasStrsm(cublasH,
					//~ HIPBLAS_SIDE_LEFT,
					//~ HIPBLAS_FILL_MODE_UPPER,
					//~ HIPBLAS_OP_N, 
					//~ HIPBLAS_DIAG_NON_UNIT,
					//~ m,
					//~ m,
					//~ &one,
					//~ &d_A[i*m*m],
					//~ m,
					//~ &d_B[i*m*m],
					//~ m);
	//~ }
	
	//~ hipEventRecord(stop, 0);
	//~ hipEventSynchronize(stop);
	//~ float time;
	//~ hipEventElapsedTime(&time, start, stop);
	
	//~ return (double) time*1e-3;
}

double diagonalise_stream_geqrf(double *d_A, double *d_W, int m, int batchSize){
}
