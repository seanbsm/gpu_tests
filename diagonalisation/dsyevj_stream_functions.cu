#include "hip/hip_runtime.h"

#include "dsyevj_stream_functions.cuh"

/* This function diagonalises by calling dsyevj in parallel using streaming */

/* This function diagonalises by calling dsyevj in parallel using streaming */
double diagonalise_stream_syevj(float *d_A, float *d_W, int m, int batchSize){
	
	
	hipsolverSyevjInfo_t syevj_params = NULL;
	/* configuration of syevj  */
	const float tol = 1.e-7;
	const int max_sweeps = 20;

	/* numerical results of syevj  */
	float residual = 0;
	int executed_sweeps = 0;
	/* step 2: configuration of syevj */
	hipsolverDnCreateSyevjInfo(&syevj_params);

	/* default value of tolerance is machine zero */
	hipsolverDnXsyevjSetTolerance(syevj_params, tol);
	
	/* default value of max. sweeps is 100 */
	hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps);
	
	
	int NBSTREAM = batchSize;
	
	hipsolverHandle_t cusolverH;
	//~ hipStream_t 	   stream	 [NBSTREAM];
	hipStream_t *stream = (hipStream_t *)malloc(NBSTREAM*sizeof(hipStream_t));
	
	int 	l_work [NBSTREAM];
	float  *d_work [NBSTREAM];
	int    *d_info [NBSTREAM];
	
	
	for (int i=0; i<NBSTREAM; i++){
		hipMalloc((void **)&d_info[i], sizeof(int));
	}
	
	for (int i=0; i<NBSTREAM; i++){
		//~ hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
		hipStreamCreate(&stream[i]);
	}
	
	hipsolverDnCreate(&cusolverH);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	for(int i=0;i<NBSTREAM;i++){
		
		hipsolverDnSsyevj_bufferSize(cusolverH,
									HIPSOLVER_EIG_MODE_VECTOR,
									HIPBLAS_FILL_MODE_UPPER,
									m,
									&d_A[i*m*m],
									m,
									&d_W[i*m],
									&l_work[i],
									syevj_params);
	
		hipMalloc((void **)&d_work[i], sizeof(float)*l_work[i]);
	}
	
	for(int i=0;i<NBSTREAM;i++){
		hipsolverSetStream(cusolverH, stream[i]);
						 
		hipsolverDnSsyevj(cusolverH,
						 HIPSOLVER_EIG_MODE_VECTOR,
						 HIPBLAS_FILL_MODE_UPPER,
						 m,
						 &d_A[i*m*m],
						 m,
						 &d_W[i*m],
						 d_work[i],
						 l_work[i],
						 d_info[i],
						 syevj_params);
	}
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	return (double) time*1e-3;
}

double diagonalise_stream_syevj(double *d_A, double *d_W, int m, int batchSize){
	
	
	hipsolverSyevjInfo_t syevj_params = NULL;
	/* configuration of syevj  */
	const double tol = 1.e-15;
	const int max_sweeps = 20;

	/* numerical results of syevj  */
	double residual = 0;
	int executed_sweeps = 0;
	/* step 2: configuration of syevj */
	hipsolverDnCreateSyevjInfo(&syevj_params);

	/* default value of tolerance is machine zero */
	hipsolverDnXsyevjSetTolerance(syevj_params, tol);
	
	/* default value of max. sweeps is 100 */
	hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps);
	
	
	int NBSTREAM = batchSize;
	
	hipsolverHandle_t cusolverH;
	//~ hipStream_t 	   stream	 [NBSTREAM];
	hipStream_t *stream = (hipStream_t *)malloc(NBSTREAM*sizeof(hipStream_t));
	
	int 	l_work [NBSTREAM];
	double *d_work [NBSTREAM];
	int    *d_info [NBSTREAM];
	
	
	for (int i=0; i<NBSTREAM; i++){
		hipMalloc((void **)&d_info[i], sizeof(int));
	}
	
	for (int i=0; i<NBSTREAM; i++){
		//~ hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
		hipStreamCreate(&stream[i]);
	}
	
	hipsolverDnCreate(&cusolverH);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	for(int i=0;i<NBSTREAM;i++){
		
		hipsolverDnDsyevj_bufferSize(cusolverH,
									HIPSOLVER_EIG_MODE_VECTOR,
									HIPBLAS_FILL_MODE_UPPER,
									m,
									&d_A[i*m*m],
									m,
									&d_W[i*m],
									&l_work[i],
									syevj_params);
	
		hipMalloc((void **)&d_work[i], sizeof(double)*l_work[i]);
	}
	
	for(int i=0;i<NBSTREAM;i++){
		hipsolverSetStream(cusolverH, stream[i]);
						 
		hipsolverDnDsyevj(cusolverH,
						 HIPSOLVER_EIG_MODE_VECTOR,
						 HIPBLAS_FILL_MODE_UPPER,
						 m,
						 &d_A[i*m*m],
						 m,
						 &d_W[i*m],
						 d_work[i],
						 l_work[i],
						 d_info[i],
						 syevj_params);
	}
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	return (double) time*1e-3;
}
