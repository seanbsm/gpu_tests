
#include "dsyevd_stream_functions.cuh"

/* This function diagonalises by calling dsyevd in parallel using streaming */
double diagonalise_stream_syevd(float  *d_A, float  *d_W, int m, int batchSize){
	
	int NBSTREAM = batchSize;
	
	hipsolverHandle_t cusolverH;
	//~ hipStream_t 	   stream	 [NBSTREAM];
	hipStream_t *stream = (hipStream_t *)malloc(NBSTREAM*sizeof(hipStream_t));
	
	int 	l_work [NBSTREAM];
	float  *d_work [NBSTREAM];
	int    *d_info [NBSTREAM];
	
	for (int i=0; i<NBSTREAM; i++){
		hipMalloc((void **)&d_info[i], sizeof(int));
	}
	
	for (int i=0; i<NBSTREAM; i++){
		//~ hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
		hipStreamCreate(&stream[i]);
	}
	
	hipsolverDnCreate(&cusolverH);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	for(int i=0;i<NBSTREAM;i++){
		hipsolverDnSsyevd_bufferSize(cusolverH,
									HIPSOLVER_EIG_MODE_VECTOR,
									HIPBLAS_FILL_MODE_UPPER,
									m,
									&d_A[i*m*m],
									m,
									&d_W[i*m],
									&l_work[i]);
	
		hipMalloc((void **)&d_work[i], sizeof(float)*l_work[i]);
	}
	
	for(int i=0;i<NBSTREAM;i++){
		hipsolverSetStream(cusolverH, stream[i]);
		
		hipsolverDnSsyevd(cusolverH,
						 HIPSOLVER_EIG_MODE_VECTOR,
						 HIPBLAS_FILL_MODE_UPPER,
						 m,
						 &d_A[i*m*m],
						 m,
						 &d_W[i*m],
						 d_work[i],
						 l_work[i],
						 d_info[i]);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	return (double) time*1e-3;
}

/* This function diagonalises by calling dsyevd in parallel using streaming */
double diagonalise_stream_syevd(double *d_A, double *d_W, int m, int batchSize){
	
	int NBSTREAM = batchSize;
	
	hipsolverHandle_t cusolverH;
	//~ hipStream_t 	   stream	 [NBSTREAM];
	hipStream_t *stream = (hipStream_t *)malloc(NBSTREAM*sizeof(hipStream_t));
	
	int 	l_work [NBSTREAM];
	double *d_work [NBSTREAM];
	int    *d_info [NBSTREAM];
	
	for (int i=0; i<NBSTREAM; i++){
		hipMalloc((void **)&d_info[i], sizeof(int));
	}
	
	for (int i=0; i<NBSTREAM; i++){
		//~ hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
		hipStreamCreate(&stream[i]);
	}
	
	hipsolverDnCreate(&cusolverH);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	for(int i=0;i<NBSTREAM;i++){
		hipsolverDnDsyevd_bufferSize(cusolverH,
									HIPSOLVER_EIG_MODE_VECTOR,
									HIPBLAS_FILL_MODE_UPPER,
									m,
									&d_A[i*m*m],
									m,
									&d_W[i*m],
									&l_work[i]);
	
		hipMalloc((void **)&d_work[i], sizeof(double)*l_work[i]);
	}
	
	for(int i=0;i<NBSTREAM;i++){
		hipsolverSetStream(cusolverH, stream[i]);
		
		hipsolverDnDsyevd(cusolverH,
						 HIPSOLVER_EIG_MODE_VECTOR,
						 HIPBLAS_FILL_MODE_UPPER,
						 m,
						 &d_A[i*m*m],
						 m,
						 &d_W[i*m],
						 d_work[i],
						 l_work[i],
						 d_info[i]);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	return (double) time*1e-3;
}
