#include "hip/hip_runtime.h"

/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include syevd_example.cpp 
 *   g++ -o a.out syevd_example.o -L/usr/local/cuda/lib64 -lcudart -lcusolver
 *
 */
#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>

//~ #include <hip/hip_runtime.h>
#include "hipblas.h" 

/* Time-keeping modules */
#include <chrono>
#include <ctime>

#include "dot.h"

void printMatrix(int m, int n, const double*A, const char* name){
	for(int row = 0 ; row < m ; row++){
		for(int col = 0 ; col < n ; col++){
			double Areg = A[row + col*m];
			printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
		}
	}
}

/* Meant for mxm matrices */
void fillMatrix(double *A, int m){
	for (int i=0; i<m; i++){
		for (int j=0; j<m; j++){
			A[i + j*m] = (i + j*j) * sqrt((double)(i + j));
		}
	}
}

void fillMatrices(double *h_A, int m, int batchSize){
	for (int b=0; b<batchSize; b++){
		fillMatrix(&h_A[b*m*m], m);
	}
}

/* This function multiplies by calling Dgemm in parallel using batch routines */
double multiplication_batch(double *d_A, double *d_B, double *d_C, int m, int batchSize){
	
	const double alpha = 1.;
	const double beta  = 0.;
	
	hipblasHandle_t cublasH;
	hipblasCreate(&cublasH);
	
	//~ auto start = std::chrono::system_clock::now();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	/* Synchronise before running multiplication scheme */
	//~ hipDeviceSynchronize();
	hipblasDgemmStridedBatched(cublasH,
							  HIPBLAS_OP_N,
							  HIPBLAS_OP_N,
							  m, m, m,
							  &alpha,
							  d_B, m,
							  m*m,
							  d_A, m,
							  m*m,
							  &beta,
							  d_C, m,
							  m*m, 
							  batchSize);
	/* Synchronise after running mulitplication scheme */
	//~ hipDeviceSynchronize();
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	//~ auto end = std::chrono::system_clock::now();
	//~ std::chrono::duration<double> time = end-start;
	
	//~ return time.count();
	return (double) time*1e-3;
}

/* This function multiplies by calling Dgemm in parallel using streaming */
double multiplication_stream(double *d_A, double *d_B, double *d_C, int m, int batchSize){
	
	int NBSTREAM = batchSize;
	
	const double alpha = 1.f;
	const double beta  = 0.f;
	
	hipblasHandle_t cublasH;
	hipblasCreate(&cublasH);
	
	hipError_t err;
	hipblasStatus_t stat;
	
	//~ hipStream_t stream [NBSTREAM];
	hipStream_t *stream = (hipStream_t *)malloc(NBSTREAM*sizeof(hipStream_t));
	for (int i=0; i<NBSTREAM; i++){
		//~ hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
		err = hipStreamCreate(&(stream[i]));
		
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	}
	
	//~ auto start = std::chrono::system_clock::now();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	/* Synchronise before running multiplication scheme */
	//~ hipDeviceSynchronize();
	for(int i=0;i<NBSTREAM;i++){
		
		hipblasSetStream(cublasH, stream[i]);
		stat = hipblasDgemm(cublasH,
						   HIPBLAS_OP_N,
						   HIPBLAS_OP_N,
						   m, m, m,
						   &alpha,
						   &d_A[i*m*m], m,
						   &d_B[i*m*m], m,
						   &beta,
						   &d_C[i*m*m], m);
					
		if(stat!=HIPBLAS_STATUS_SUCCESS){printf("error code %d, line(%d)\n", stat, __LINE__);exit(EXIT_FAILURE);}
	}
	/* Synchronise after running multiplication scheme */
	//~ hipDeviceSynchronize();
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	//~ auto end = std::chrono::system_clock::now();
	//~ std::chrono::duration<double> time = end-start;
	
	//~ return time.count();
	return (double) time*1e-3;
}

int main(int argc, char*argv[]){
	
	const int m = 4*32;
	const int batchSize = 2*32;

	/* Declare host arrays */
	double *A = new double [m*m*batchSize];
	double *B = new double [m*m*batchSize];
	double *C = new double [m*m*batchSize];
	
	/* Declare device arrays */
	double *d_A  = NULL; /* m-by-m-by-batchSize */
	double *d_B  = NULL; /* m-by-m-by-batchSize */
	double *d_C  = NULL; /* m-by-m-by-batchSize */
	
	/* Fill up array A and B with matrix elements */
	fillMatrices(A, m, batchSize);
	fillMatrices(B, m, batchSize);
	
	/* Allocate A, B, and C on device */
	hipMalloc ((void**)&d_A, sizeof(double)*m*m*batchSize);
	hipMalloc ((void**)&d_B, sizeof(double)*m*m*batchSize);
	hipMalloc ((void**)&d_C, sizeof(double)*m*m*batchSize);
	
	/* Copy A to device */
	hipMemcpy(d_A, A, sizeof(double)*m*m*batchSize, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(double)*m*m*batchSize, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	 
	/* Call multiplication routine of choice */
	//~ double time_gpu = multiplication_stream(d_A, d_B, d_C, m, batchSize);
	double time_gpu = multiplication_batch(d_A, d_B, d_C, m, batchSize);
	
	std::cout<<"Time gpu: "<< time_gpu << " s" << std::endl;
	
	/* Code C from device */
	hipMemcpy(C, d_C, sizeof(double) * m * m * batchSize, hipMemcpyDeviceToHost);
	
	/* CPU BENCHMARKING */
	
	/* Declare CPU arrays */
	double *A_CPU = new double [m*m*batchSize];
	double *B_CPU = new double [m*m*batchSize];
	double *C_CPU = new double [m*m*batchSize];
	
	/* Fill up array A and B with matrix elements */
	fillMatrices(A_CPU, m, batchSize);
	fillMatrices(B_CPU, m, batchSize);
	
	auto start = std::chrono::system_clock::now();
	
	/* Multipli A and B using MKL LAPACK */
	/* Arg 1 is A */
	/* Arg 1 is B */
	/* Arg 1 is C */
	/* Arg 4, 5, and 6 are the dimensions (all m) */
	for (int i=0; i<batchSize; i++){
		dot_MM(&A_CPU[i*m*m], &B_CPU[i*m*m], &C_CPU[i*m*m], m, m, m);
	}
	
	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> time_raw = end-start;
	double time_cpu = time_raw.count();
	
	std::cout<<"Time cpu: "<< time_cpu << " s" << std::endl;
	
	//~ for (int M=0; M<batchSize; M++){
		//~ for (int i=0; i<m; i++){
			//~ for (int j=0; j<m; j++){
				//~ std::cout << V_CPU[M*m*m + i*m+j] << std::endl;
			//~ }
		//~ }
	//~ }
	
	/* Print any numerically big differences in eigenvalues between GPU and CPU*/
	double maxDiff = 0;
	for (int M=0; M<batchSize; M++){
		for (int i=0; i<m*m; i++){
			double diff = abs(C_CPU[M*m*m + i] - C[M*m*m + i]);
			if (diff > maxDiff){
				maxDiff = diff;
				//~ std::cout << M << " " << i << " " << diff << std::endl;
			}
		}
	}
	
	std::cout << std::endl;
	std::cout << "Max diff: " << maxDiff << std::endl;

	/* free resources */
	if (d_A) hipFree(d_A);
	if (d_B) hipFree(d_B);
	if (d_C) hipFree(d_C);

	delete [] A;
	delete [] B;
	delete [] C;

	delete [] A_CPU;
	delete [] B_CPU;
	delete [] C_CPU;

	hipDeviceReset();

	return 0;
}

